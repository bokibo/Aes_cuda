#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "consts.h"
#include <malloc.h>	
#include <fstream>
#include <stdlib.h>
#include <iostream>
#include <time.h>

using namespace std;
//macros multiplication for Inverse MixColumns
#define xtime(x)   ((x<<1) ^ (((x>>7) & 1) * 0x1b))
#define Multiply(x,y) (((y & 1) * x) ^ ((y>>1 & 1) * xtime(x)) ^ ((y>>2 & 1) * xtime(xtime(x))) ^ ((y>>3 & 1) * xtime(xtime(xtime(x)))) ^ ((y>>4 & 1) * xtime(xtime(xtime(xtime(x))))))

__constant__ unsigned char Sbox_dev[256] =
{
	0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76 ,
	0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0 ,
	0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15 ,
	0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75 ,
	0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84 ,
	0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF ,
	0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8 ,
	0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2 ,
	0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73 ,
	0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB ,
	0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79 ,
	0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08 ,
	0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A ,
	0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E ,
	0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF ,
	0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
};

__constant__ unsigned char InvSbox_dev[256] =
{
	0x52, 0x09, 0x6A, 0xD5, 0x30, 0x36, 0xA5, 0x38, 0xBF, 0x40, 0xA3, 0x9E, 0x81, 0xF3, 0xD7, 0xFB ,
	0x7C, 0xE3, 0x39, 0x82, 0x9B, 0x2F, 0xFF, 0x87, 0x34, 0x8E, 0x43, 0x44, 0xC4, 0xDE, 0xE9, 0xCB ,
	0x54, 0x7B, 0x94, 0x32, 0xA6, 0xC2, 0x23, 0x3D, 0xEE, 0x4C, 0x95, 0x0B, 0x42, 0xFA, 0xC3, 0x4E ,
	0x08, 0x2E, 0xA1, 0x66, 0x28, 0xD9, 0x24, 0xB2, 0x76, 0x5B, 0xA2, 0x49, 0x6D, 0x8B, 0xD1, 0x25 ,
	0x72, 0xF8, 0xF6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xD4, 0xA4, 0x5C, 0xCC, 0x5D, 0x65, 0xB6, 0x92 ,
	0x6C, 0x70, 0x48, 0x50, 0xFD, 0xED, 0xB9, 0xDA, 0x5E, 0x15, 0x46, 0x57, 0xA7, 0x8D, 0x9D, 0x84 ,
	0x90, 0xD8, 0xAB, 0x00, 0x8C, 0xBC, 0xD3, 0x0A, 0xF7, 0xE4, 0x58, 0x05, 0xB8, 0xB3, 0x45, 0x06 ,
	0xD0, 0x2C, 0x1E, 0x8F, 0xCA, 0x3F, 0x0F, 0x02, 0xC1, 0xAF, 0xBD, 0x03, 0x01, 0x13, 0x8A, 0x6B ,
	0x3A, 0x91, 0x11, 0x41, 0x4F, 0x67, 0xDC, 0xEA, 0x97, 0xF2, 0xCF, 0xCE, 0xF0, 0xB4, 0xE6, 0x73 ,
	0x96, 0xAC, 0x74, 0x22, 0xE7, 0xAD, 0x35, 0x85, 0xE2, 0xF9, 0x37, 0xE8, 0x1C, 0x75, 0xDF, 0x6E ,
	0x47, 0xF1, 0x1A, 0x71, 0x1D, 0x29, 0xC5, 0x89, 0x6F, 0xB7, 0x62, 0x0E, 0xAA, 0x18, 0xBE, 0x1B ,
	0xFC, 0x56, 0x3E, 0x4B, 0xC6, 0xD2, 0x79, 0x20, 0x9A, 0xDB, 0xC0, 0xFE, 0x78, 0xCD, 0x5A, 0xF4 ,
	0x1F, 0xDD, 0xA8, 0x33, 0x88, 0x07, 0xC7, 0x31, 0xB1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xEC, 0x5F ,
	0x60, 0x51, 0x7F, 0xA9, 0x19, 0xB5, 0x4A, 0x0D, 0x2D, 0xE5, 0x7A, 0x9F, 0x93, 0xC9, 0x9C, 0xEF ,
	0xA0, 0xE0, 0x3B, 0x4D, 0xAE, 0x2A, 0xF5, 0xB0, 0xC8, 0xEB, 0xBB, 0x3C, 0x83, 0x53, 0x99, 0x61 ,
	0x17, 0x2B, 0x04, 0x7E, 0xBA, 0x77, 0xD6, 0x26, 0xE1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0C, 0x7D
};

__constant__ unsigned char MixCol_dev[4][4] =
{
	{ 0x02,0x03,0x01,0x01 },
	{ 0x01,0x02,0x03,0x01 },
	{ 0x01,0x01,0x02,0x03 },
	{ 0x03,0x01,0x01,0x02 }
};

__constant__ unsigned char InvMixCol_dev[4][4] = {
	{ 0x0e, 0x0b, 0x0d, 0x09 },
	{ 0x09, 0x0e, 0x0b, 0x0d },
	{ 0x0d, 0x09, 0x0e, 0x0b },
	{ 0x0b, 0x0d, 0x09, 0x0e }
};


//----------------------------------------
typedef struct {
	unsigned char item[4][4];
} Block;


//----------------------------------------
// file length in number of characters
__host__ long file_length(const char* filename) {
	FILE * f = fopen(filename, "r");
	long length;
	if (f)
	{
		fseek(f, 0, SEEK_END);
		length = ftell(f);
		fclose(f);
		return length;
	}
	else
		return 0;
}
//----------------------------------------
// KEY SCHEDULING ALGORITHM
__host__ void key_scheduling(Block * keys) {
	
	//initial key
	unsigned char key[4][4] = {
		{ 0x54, 0x73, 0x20, 0x67 },
		{ 0x68, 0x20, 0x4b, 0x20 },
		{ 0x61, 0x6d, 0x75, 0x46 },
		{ 0x74, 0x79, 0x6e, 0x75 } };

	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			keys[0].item[i][j] = key[i][j];
		}
	}

	// key scheduling algorithm
	for (int k = 1; k <= 10; k++) {
		Block tempNew;
		Block tempOld = keys[k - 1];
		unsigned int temp[4] = { 
			tempOld.item[0][3], 
			tempOld.item[1][3], 
			tempOld.item[2][3], 
			tempOld.item[3][3] 
		};
	// ROTWORD
		unsigned int t;
		t = temp[0];
		temp[0] = temp[1];
		temp[1] = temp[2];
		temp[2] = temp[3];
		temp[3] = t;

		//SUBBYTES
		for (int i = 0; i < 4; i++) {
			temp[i] = Sbox1[temp[i]];
		}
		unsigned int temp2[4] = { 
			tempOld.item[0][0], 
			tempOld.item[1][0], 
			tempOld.item[2][0], 
			tempOld.item[3][0] };

		//RCON	
		//xor second column and temp and Rcon 1st round
		for (int i = 0; i < 4; i++) {
			temp2[i] = temp[i] ^ tempOld.item[i][0];
			temp2[i] = temp2[i] ^ Rcon[i][k - 1];
		}
		for (int i = 0; i < 4; i++)  //first column of 2nd key
			tempNew.item[i][0] = temp2[i];
		for (int j = 1; j < 4; j++) {
			for (int i = 0; i < 4; i++)
			{
				tempNew.item[i][j] = (tempNew.item[i][j - 1] ^ tempOld.item[i][j]);
			}
		}
		keys[k] = tempNew;
	} //end of key scheduling
}
//----------------------------------------

__host__ void real_initialization(Block * in, Block * out, int real_num_of_blocks) {
	for (size_t t = 0; t < real_num_of_blocks; t++)
	{
		for (size_t i = 0; i < 4; i++)
		{
			for (size_t j = 0; j < 4; j++)
			{
				out[t].item[i][j] = in[t].item[i][j];
			}
		}
	}
}

//----------------------------------------
// reading text file and initialization of array of blocks
__host__ int  initialization(char* source, Block * plaintext, int num_of_blocks) {
	Block * temp;
	int num = 0;
	hipHostMalloc((void**)&temp, num_of_blocks * sizeof(Block));
	ifstream ifs(source);
	int k = 0;
	while (ifs) {
		ifs.read((char *)temp[k].item, 16);
		k++;
	}
	for (size_t t = 0; t < num_of_blocks; t++)
	{
		for (size_t i = 0; i < 4; i++)
		{
			for (size_t j = 0; j < 4; j++)
			{
				if (temp[t].item[i][j] == '\n')
					num++;
				plaintext[t].item[j][i] = temp[t].item[i][j];
			}
		}
	}
	hipFree(temp);
	return num;
}
//----------------------------------------
// printing blocks to stdout
__host__ void printBlocks(Block * in, int num_of_blocks) {
	for (size_t t = 0; t < num_of_blocks; t++)
	{
		fprintf(stdout, "%d. block\n", t);
		for (size_t i = 0; i < 4; i++)
		{
			for (size_t j = 0; j < 4; j++)
			{
				fprintf(stdout, " %0x", in[t].item[i][j]);
			}
			fprintf(stdout, "\n");
		}
		fprintf(stdout, "-----------------\n");
	}
}
//----------------------------------------
// writing array of blocks into text file
__host__ void writeToFile(Block * in, int num_of_blocks, char * filename) {
	FILE * file = fopen(filename, "w");
	if (file) {
		for (size_t t = 0; t < num_of_blocks; t++)
		{
			for (size_t i = 0; i < 4; i++)
			{
				for (size_t j = 0; j < 4; j++)
				{
					if (in[t].item[j][i] != 0)
						fprintf(file, "%c", in[t].item[j][i]);
				}
			}
		}
	}
	else
		fprintf(stdout, "Error opening file %s ", filename);
}

// new mix columns function
__device__ unsigned int mc(unsigned int a, unsigned int b, unsigned int c, unsigned int d)
{
	unsigned int Tmp, Tm, e;
	Tmp = a ^ b ^ c ^ d;
	Tm = a ^ b;
	Tm = xtime(Tm);
	e = Tm ^ Tmp ^ a;
	return e;
}
//----------------------------------------
// mix columns 
__device__ unsigned int mixColumns(unsigned int m0,
	unsigned int m1,
	unsigned int m2,
	unsigned int m3,
	unsigned int c0,
	unsigned int c1,
	unsigned int c2,
	unsigned int c3) {
	unsigned int rez0 = 0;
	unsigned int rez1 = 0;
	unsigned int rez2 = 0;
	unsigned int rez3 = 0;

	switch (m0)
	{
	case 1:
		rez0 = c0;
		break;
	case 2:
		rez0 = c0 << 1;
		if ((((c0 & 0x80) >> 7) & 0x01) == 1)
			rez0 ^= 0x1b;
		break;
	case 3:
		rez0 = c0 << 1;
		if ((((c0 & 0x80) >> 7) & 0x01) == 1)
			rez0 ^= 0x1b;
		rez0 ^= c0;
		break;
	default:
		break;
	}
	switch (m1)
	{
	case 1:
		rez1 = c1;
		break;
	case 2:
		rez1 = c1 << 1;
		if ((((c1 & 0x80) >> 7) & 0x01) == 1)
			rez1 ^= 0x1b;
		break;
	case 3:
		rez1 = c1 << 1;
		if ((((c1 & 0x80) >> 7) & 0x01) == 1)
			rez1 ^= 0x1b;
		rez1 ^= c1;
		break;
	default:
		break;
	}
	switch (m2)
	{
	case 1:
		rez2 = c2;
		break;
	case 2:
		rez2 = c2 << 1;
		if ((((c2 & 0x80) >> 7) & 0x01) == 1)
			rez2 ^= 0x1b;
		break;
	case 3:
		rez2 = c2 << 1;
		if ((((c2 & 0x80) >> 7) & 0x01) == 1)
			rez2 ^= 0x1b;
		rez2 ^= c2;
		break;
	default:
		break;
	}
	switch (m3)
	{
	case 1:
		rez3 = c3;
		break;
	case 2:
		rez3 = c3 << 1;
		if ((((c3 & 0x80) >> 7) & 0x01) == 1)
			rez3 ^= 0x1b;
		break;
	case 3:
		rez3 = c3 << 1;
		if ((((c3 & 0x80) >> 7) & 0x01) == 1)
			rez3 ^= 0x1b;
		rez3 ^= c3;
		break;
	default:
		break;
	}
	return rez0 ^ rez1 ^ rez2 ^ rez3;
}
//----------------------------------------
// inverse mix columns
__device__ unsigned int inverseMixColumns(unsigned int in0,
	unsigned int in1,
	unsigned int in2,
	unsigned int in3,
	unsigned int p0,
	unsigned int p1,
	unsigned int p2,
	unsigned int p3) {
	return Multiply(p0, in0) ^ Multiply(p1, in1) ^ Multiply(p2, in2) ^ Multiply(p3, in3);
}
//----------------------------------------
// encryption kernel
__global__ void encrypt(Block *keys, Block *plaintext, Block *ciphertext, unsigned int num_of_blocks)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	int blockNumber = blockIdx.y;

	//initial add round key
	ciphertext[blockNumber].item[i][j] = keys[0].item[i][j] ^ plaintext[blockNumber].item[i][j];
	//__syncthreads();

	for (int k = 1; k < 10; k++)
	{
		//subBytes
		ciphertext[blockNumber].item[i][j] = Sbox_dev[ciphertext[blockNumber].item[i][j]];
		//__syncthreads();

		//shift rows
		ciphertext[blockNumber].item[i][j] = ciphertext[blockNumber].item[i][(j + i) % 4];
		//__syncthreads();

		//mixColumns
		ciphertext[blockNumber].item[i][j] = mc(ciphertext[blockNumber].item[i][j],
			ciphertext[blockNumber].item[(i+1)%4][j],
			ciphertext[blockNumber].item[(i+2)%4][j],
			ciphertext[blockNumber].item[(i+3)%4][j]);

		//add round key
		ciphertext[blockNumber].item[i][j] = keys[k].item[i][j] ^ ciphertext[blockNumber].item[i][j];
		//__syncthreads();
	}
	//subbytes
	ciphertext[blockNumber].item[i][j] = Sbox_dev[ciphertext[blockNumber].item[i][j]];
	//__syncthreads();

	//rotwords
	ciphertext[blockNumber].item[i][j] = ciphertext[blockNumber].item[i][(j + i) % 4];
	//__syncthreads();

	//add round key
	ciphertext[blockNumber].item[i][j] = keys[10].item[i][j] ^ ciphertext[blockNumber].item[i][j];
	//__syncthreads();
}
//----------------------------------------
// decryption kernel
__global__ void decrypt(Block *keys, Block *plaintext, Block *ciphertext, unsigned int num_of_blocks) {

	int i = threadIdx.x;
	int j = threadIdx.y;
	int blockNumber = blockIdx.y;

	//inverse add round key
	plaintext[blockNumber].item[i][j] = ciphertext[blockNumber].item[i][j] ^ keys[10].item[i][j];
	//__syncthreads();

	for (size_t k = 9; k >= 1; k--)
	{
		//inverse shift rows
		plaintext[blockNumber].item[i][j] = plaintext[blockNumber].item[i][(4 + j - i) % 4];
		//__syncthreads();

		//inverse subbytes
		plaintext[blockNumber].item[i][j] = InvSbox_dev[plaintext[blockNumber].item[i][j]];
		//__syncthreads();

		//inverse add round key
		plaintext[blockNumber].item[i][j] = plaintext[blockNumber].item[i][j] ^ keys[k].item[i][j];
		//__syncthreads();


		//inverse mixColumns
		plaintext[blockNumber].item[i][j] = inverseMixColumns(
			InvMixCol_dev[i][0],
			InvMixCol_dev[i][1],
			InvMixCol_dev[i][2],
			InvMixCol_dev[i][3],
			plaintext[blockNumber].item[0][j],
			plaintext[blockNumber].item[1][j],
			plaintext[blockNumber].item[2][j],
			plaintext[blockNumber].item[3][j]);
		//__syncthreads();


	}
	//inverse shift rows
	plaintext[blockNumber].item[i][j] = plaintext[blockNumber].item[i][(4 + j - i) % 4];
	//__syncthreads();

	//inverse subbytes
	plaintext[blockNumber].item[i][j] = InvSbox_dev[plaintext[blockNumber].item[i][j]];
	//__syncthreads();

	//inverse add round key
	plaintext[blockNumber].item[i][j] = plaintext[blockNumber].item[i][j] ^ keys[0].item[i][j];
	//__syncthreads();
}
//----------------------------------------
// main function
int main()
{
	hipDeviceProp_t deviceProp;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	//----------------------- device properties ------------------------------------
	cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "cudaGetProp failed!");
	}
	fprintf(stdout, "   Hardware information:\n");
	fprintf(stdout, "     Device name:                  %s\n", deviceProp.name);
	fprintf(stdout, "     Major revision number:        %d\n", deviceProp.major);
	fprintf(stdout, "     Minor revision Number:        %d\n", deviceProp.minor);
	fprintf(stdout, "     Memory clock rate:            %d  MHz\n", deviceProp.memoryClockRate / 1000);
	fprintf(stdout, "     Clock Rate:                   %d MHz\n", deviceProp.clockRate / 1000);
	fprintf(stdout, "     Total Global Memory:          %d MB\n", deviceProp.totalGlobalMem / 1024 / 1024);
	fprintf(stdout, "     L2 Cache memory size:         %d  KB\n", deviceProp.l2CacheSize / 1024);
	fprintf(stdout, "     Total shared mem per block:   %d   KB\n", deviceProp.sharedMemPerBlock / 1024);
	fprintf(stdout, "     Total const mem size:         %d   KB\n", deviceProp.totalConstMem / 1024);
	fprintf(stdout, "     Warp size:                    %d\n", deviceProp.warpSize);
	fprintf(stdout, "     Maximum block dimensions:     %d x %d x %d\n", deviceProp.maxThreadsDim[0],
		deviceProp.maxThreadsDim[1],
		deviceProp.maxThreadsDim[2]);
	fprintf(stdout, "     Maximum grid dimensions:      %d x %d x %d\n", deviceProp.maxGridSize[0],
		deviceProp.maxGridSize[1],
		deviceProp.maxGridSize[2]);
	fprintf(stdout, "     Number of muliprocessors:     %d\n", deviceProp.multiProcessorCount);
	fprintf(stdout, "     Max threads per block:        %d\n", deviceProp.maxThreadsPerBlock);
	fprintf(stdout, "     Supports conncurent kernels:  %s\n\n\n", (deviceProp.concurrentKernels == 1) ? "Yes" : "No");

	ofstream off3;
	Block * keys;
	cudaStatus = hipHostMalloc((void**)&keys, 11 * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Keys allocation error #%d\n", cudaStatus);
	}
	key_scheduling(keys);

	fprintf(stdout, "   Application information\n");
	cout << "     Enter text file name: ";
	char name[20];
	scanf("%s", name);

	long plaintext_length = file_length(name);
	long num_of_blocks = (plaintext_length % 16 == 0) ? plaintext_length / 16 : plaintext_length / 16 + 1;

	fprintf(stdout, "     Plaintext length: %ld characters \n", plaintext_length);

	Block * plaintext;
	cudaStatus = hipHostMalloc((void**)&plaintext, num_of_blocks * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Plaintext allocation error #%d\n", cudaStatus);
	}
	int num_of_zeros = initialization(name, plaintext, num_of_blocks);
	int real_num_of_block = ((plaintext_length - num_of_zeros) % 16 == 0) ? (plaintext_length - num_of_zeros) / 16 : (plaintext_length - num_of_zeros) / 16 + 1;
	fprintf(stdout, "     Number of blocks: %d \n", num_of_blocks);
	fprintf(stdout, "     Number of real blocks: %d \n", real_num_of_block);

	Block * real_plaintext;
	cudaStatus = hipHostMalloc((void**)&real_plaintext, real_num_of_block * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Real plaintext allocation error #%d\n", cudaStatus);
	}
	hipMemcpy(real_plaintext, plaintext, real_num_of_block * sizeof(Block), hipMemcpyHostToHost);
	hipFree(plaintext);

	Block * ciphertext;
	cudaStatus = hipHostMalloc((void**)&ciphertext, real_num_of_block * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Ciphertext allocation error #%d\n", cudaStatus);
	}
	Block * plaintext2;
	cudaStatus = hipHostMalloc((void**)&plaintext2, real_num_of_block * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Plaintext2 allocation error #%d\n", cudaStatus);
	}

	//--------------- device memory allocation -------------------
	hipEvent_t startAlloc, stopAlloc;
	float timeAlloc;
	hipEventCreate(&startAlloc);
	hipEventCreate(&stopAlloc);
	hipEventRecord(startAlloc, 0);

	Block *keys_dev;
	cudaStatus = hipMalloc((void**)&keys_dev, 11 * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Allocating memory for key blocks failed!");
		goto Error;
	}
	cudaStatus = hipMemset(keys_dev, 0, 11 * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "keys memset error #%d\n", cudaStatus);
	}

	Block *plaintext_dev;
	cudaStatus = hipMalloc((void**)&plaintext_dev, real_num_of_block * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Allocating memory for plaintext blocks failed!");
		goto Error;
	}

	cudaStatus = hipMemset(plaintext_dev, 0, real_num_of_block * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "plaintext memset error #%d\n", cudaStatus);
	}

	Block *ciphertext_dev;
	cudaStatus = hipMalloc((void**)&ciphertext_dev, real_num_of_block * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Allocating memory for ciphertext blocks failed!");
		goto Error;
	}

	cudaStatus = hipMemset(ciphertext_dev, 0, real_num_of_block * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "cipher memset error #%d\n", cudaStatus);
	}

	Block* plaintext2_dev;
	cudaStatus = hipMalloc((void**)&plaintext2_dev, real_num_of_block * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Allocating memory for decrypted text blocks failed!");
		goto Error;
	}

	cudaStatus = hipMemset(plaintext2_dev, 0, real_num_of_block * sizeof(Block));
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "plaintext2 memset error #%d\n", cudaStatus);
	}
	//------------------ copying block from host to device -----------------
	cudaStatus = hipMemcpy(keys_dev, keys, 11 * sizeof(Block), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Copying key blocks on device failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(plaintext_dev, real_plaintext, real_num_of_block * sizeof(Block), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Copying plaintext blocks on device failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(ciphertext_dev, ciphertext, real_num_of_block * sizeof(Block), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Copying ciphertext blocks on device failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(plaintext2_dev, plaintext2, real_num_of_block * sizeof(Block), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Copying decrypted blocks on device failed!");
		goto Error;
	}
	hipEventRecord(stopAlloc, 0);
	hipEventSynchronize(stopAlloc);
	hipEventElapsedTime(&timeAlloc, startAlloc, stopAlloc);
	
	//----------------------------------------------------------------------
	hipFree(keys);
	hipFree(real_plaintext);

	int kernel_rounds = real_num_of_block / 50000;
	int rest = real_num_of_block - (kernel_rounds * 50000);

	dim3 threadsPerBlock(4, 4);
	dim3 numBlocks(1, 50000);
	dim3 numBlocks2(1, rest);
	fprintf(stdout, "\n   Encryption.........\n");
	hipEvent_t startEnc, stopEnc;
	float timeEnc;
	hipEventCreate(&startEnc);
	hipEventCreate(&stopEnc);
//	int num_of_rounds;
	hipEventRecord(startEnc, 0);
	for (size_t i = 0; i < kernel_rounds; i++)
	{
		encrypt << <numBlocks, threadsPerBlock >> > (keys_dev, plaintext_dev+i*50000, ciphertext_dev+i* 50000, 50000);
	}
	encrypt << <numBlocks2, threadsPerBlock >> > (keys_dev, plaintext_dev + kernel_rounds * 50000, ciphertext_dev + kernel_rounds * 50000, rest);
	hipEventRecord(stopEnc, 0);
	hipEventSynchronize(stopEnc);
	hipEventElapsedTime(&timeEnc, startEnc, stopEnc);
	fprintf(stdout, "     Encryption time %.2f ms\n\n", timeEnc);

	//------------------ decryption --------------------------------------
	fprintf(stdout, "   Decryption.......\n");
	hipEvent_t startDec, stopDec;
	float timeDec;
	hipEventCreate(&startDec);
	hipEventCreate(&stopDec);
	hipEventRecord(startDec, 0);

	for (size_t i = 0; i < kernel_rounds; i++)
	{
		decrypt << <numBlocks, threadsPerBlock >> > (keys_dev, plaintext2_dev + i * 50000, ciphertext_dev + i * 50000, 50000);
	}
	decrypt << <numBlocks2, threadsPerBlock >> > (keys_dev, plaintext2_dev + kernel_rounds * 50000, ciphertext_dev + kernel_rounds * 50000, rest);
	hipEventRecord(stopDec, 0);
	hipEventSynchronize(stopDec);
	hipEventElapsedTime(&timeDec, startDec, stopDec);
	fprintf(stdout, "     Decryption time %.2f ms\n", timeDec);
	cudaStatus = hipMemcpy(ciphertext, ciphertext_dev, real_num_of_block * sizeof(Block), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Copying ciphertext blocks on host failed!");
		goto Error;
	}
	hipEvent_t startDealloc, stopDealloc;
	float timeDealloc;
	hipEventCreate(&startDealloc);
	hipEventCreate(&stopDealloc);
	hipEventRecord(startDealloc, 0);
	cudaStatus = hipMemcpy(plaintext2, plaintext2_dev, real_num_of_block * sizeof(Block), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stdout, "Copying decrypted blocks on host failed!");
		goto Error;
	}
	hipEventRecord(stopDealloc, 0);
	hipEventSynchronize(stopDealloc);
	hipEventElapsedTime(&timeDealloc, startDealloc, stopDealloc);

	int index = 0;
	for (size_t i = 0; i < 20; i++)
	{
		if (name[i] == '.')
		{
			index = i;
			break;
		}
	}
	
	name[index] = '_';
	name[index + 1] = 'i';
	name[index + 2] = 'n';
	name[index + 3] = 'f';
	name[index + 4] = 'o';
	name[index + 5] = '.';
	name[index + 6] = 't';
	name[index + 7] = 'x';
	name[index + 8] = 't';
	name[index + 9] = '\0';
	off3.open(name, ofstream::trunc);
	off3 << "Number of characters in file: " << plaintext_length << endl;
	off3 << "Number of blocks with zeros: " << num_of_blocks << endl;
	off3 << "Number of zeros: " << num_of_zeros << endl;
	off3 << "Number of blocks without zeros: " << real_num_of_block << endl;
	off3 << "Encryption finished in " << timeEnc << " miliseconds" << endl;
	off3 << "Decryption finished in " << timeDec << " miliseconds" << endl;
	off3.close();

	fprintf(stdout, "\n   Writing to files....\n\n");
	//writeToFile(ciphertext, real_num_of_block, "encrypted_text.txt");
	//writeToFile(plaintext2, real_num_of_block, "decrypted_text.txt");
Error:
	hipEvent_t startDealloc2, stopDealloc2;
	float timeDealloc2;
	hipEventCreate(&startDealloc2);
	hipEventCreate(&stopDealloc2);
	hipEventRecord(startDealloc2, 0);

	hipFree(ciphertext);
	hipFree(plaintext2);
	hipFree(keys_dev);
	hipFree(plaintext_dev);
	hipFree(ciphertext_dev);

	hipEventRecord(stopDealloc2, 0);
	hipEventSynchronize(stopDealloc2);
	hipEventElapsedTime(&timeDealloc2, startDealloc2, stopDealloc2);

	//fprintf(stdout, "\nPress enter to end......");
	fprintf(stdout, "\n\nAllocation and copying time %.2f ms\n", timeAlloc);
	fprintf(stdout, "Deallocation and copying time %.2f ms\n\n", timeDealloc + timeDealloc2);
	return 0;
}